#include "markov_motion_update/motion_update.h"
#include "ros/ros.h"
#include <vector>
#include <numeric>
#define IDX2M(x,y,d2) ((x)*(d2)+(y))
#define IDX2G(a,x,y,d2,d3) (((a)*(d2)*(d3))+(IDX2M(x,y,d3)))
#define IDX2Q(opa,a,x,y,d2,d3,d4) (((opa)*(d2)*(d3)*(d4))+(a)*(d3)*(d4)+(x)*(d4)+(y))

double CPUdot(double*& A, double*& B, int size)
{
  double product = 0.0;
  for(int i = 0 ; i < size ; ++i)
    product += A[i] * B[i];
  return product;
}

int check(double*& vec, double*& bak, int size)
{
  for(int i = 0 ; i < size ; ++i)
  {
    if(vec[i] != bak[i])
      return 1;
  }
  return 0;
}

void showMemInfo()
{
        // show memory usage of GPU

  size_t free_byte ;
  size_t total_byte ;
  
  hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
  
  if ( hipSuccess != cuda_status )
  {
    printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
    exit(1);
  }
  double free_db = (double)free_byte ;
  double total_db = (double)total_byte ;
  double used_db = total_db - free_db ;
  printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n", used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
}

int cublasFunc(void)
{
  ros::Time::init();
  std::vector<int> vec_cuda1;
  std::vector<int> vec_cuda2;
  std::vector<int> vec_cuda3;
  std::vector<int> vec_cudaall;
  std::vector<int> vec_cpu;
  hipError_t cudaStat;
  hipblasStatus_t stat;
  hipblasHandle_t handle;
  int map_x = 700;
  int map_y = 500;
  int map_a = 72;
  int mask_side = 91;//note that this has to be an odd positive number
  int total_mem_allocated = 0;//GPU memory size allocated by this program in Byte
  //prerequisite
  double* pre_w = 0;//previous weight vector, map_a*map_x*map_y
  double* cur_w = 0;//previous weight vector, map_a*map_x*map_y
  double* single_w = 0;
  pre_w = (double *)malloc( map_x*map_y*map_a*sizeof(*pre_w));
  cur_w = (double *)malloc( map_x*map_y*map_a*sizeof(*cur_w));
  single_w = (double *)malloc( sizeof(*single_w));
  *single_w = 0;
  if(!pre_w || !cur_w)
  {
    printf("host memory allocation failed\n");
    return 1;
  }
  int i,j,k;
  for (k = 0; k < map_a; k++)
    for (j = 0; j < map_y; j++)
      for (i = 0; i < map_x; i++)
      {
        pre_w[IDX2G(k,i,j,map_x,map_y)] = (double)IDX2G(k,i,j,map_x,map_y);
        cur_w[IDX2G(k,i,j,map_x,map_y)] = 0;
      }
  //printf("pre_w\n");
  //printWG(pre_w, map_x, map_y, map_a);

  //host memory pointer
  double* mask_w_mat = 0;
  size_t ngb_pre_w_vec_size = map_a*mask_side*mask_side;
  size_t mask_w_mat_size = map_a*ngb_pre_w_vec_size;
  if(allocateMaskWMat(mask_w_mat, mask_side, map_a, mask_w_mat_size)!=0)
    return EXIT_FAILURE;
  double* ngb_pre_w_vec = 0;
  double* ngb_pre_w_vec_bak = 0;
  double* mask_w_vec_bak = 0;
  if(allocateNgbPreWVec(ngb_pre_w_vec, mask_side, map_a, ngb_pre_w_vec_size)!=0)
    return EXIT_FAILURE;
  if(allocateNgbPreWVec(ngb_pre_w_vec_bak, mask_side, map_a, ngb_pre_w_vec_size)!=0)
    return EXIT_FAILURE;
  if(allocateNgbPreWVec(mask_w_vec_bak, mask_side, map_a, ngb_pre_w_vec_size)!=0)
    return EXIT_FAILURE;
  //device memory pointer
  double* mask_w_mat_dev;//mask weight matrix, map_a by map_a*mask_side*mask_side
  double* ngb_pre_w_vec_dev;//neighbors' previous weight vector, map_a*mask_side*mask_side
  double* cur_w_dev = 0;
  showMemInfo();
  cudaStat = hipMalloc ((void **)(&mask_w_mat_dev), mask_w_mat_size*sizeof(*mask_w_mat));
  total_mem_allocated += mask_w_mat_size*sizeof(*mask_w_mat);
  if (cudaStat != hipSuccess)
  {
      printf ("device memory allocation failed\n");
      hipFree (mask_w_mat_dev);
      return EXIT_FAILURE;
  }
  cudaStat = hipMalloc ((void **)(&ngb_pre_w_vec_dev), ngb_pre_w_vec_size*sizeof(*ngb_pre_w_vec));
  total_mem_allocated += ngb_pre_w_vec_size*sizeof(*ngb_pre_w_vec);
  if (cudaStat != hipSuccess)
  {
      printf ("device memory allocation failed\n");
      hipFree (ngb_pre_w_vec_dev);
      return EXIT_FAILURE;
  }
  cudaStat = hipMalloc ((void **)(&cur_w_dev), map_x*map_y*map_a*sizeof(*cur_w));
  total_mem_allocated += map_x*map_y*map_a*sizeof(*cur_w);
  if (cudaStat != hipSuccess)
  {
      printf ("device memory allocation failed\n");
      hipFree (cur_w_dev);
      return EXIT_FAILURE;
  }
  printf("allocated %d Byte of GPU memory\n",total_mem_allocated);
  //cublas
  stat = hipblasCreate(&handle);
  if (stat != HIPBLAS_STATUS_SUCCESS)
  {
      printf ("CUBLAS initialization failed\n");
      return EXIT_FAILURE;
  }
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
  printf ("CUBLAS initialization succeed\n");
  //printf("original cur_w\n");
  //printWG(cur_w, map_x, map_y, map_a);
  stat = hipblasSetVector (map_x*map_y*map_a , sizeof(*cur_w_dev), cur_w, 1, cur_w_dev, 1);
  if (stat != HIPBLAS_STATUS_SUCCESS)
  {
      printf ("hipblasSetVector failed, %d\n",stat);
      return EXIT_FAILURE;
  }
  hipblasGetVector(map_x*map_y*map_a, sizeof(*cur_w_dev), cur_w_dev, 1, cur_w, 1);
  //printf("cur_w_dev\n");
  //printWG(cur_w, map_x, map_y, map_a);
  stat = hipblasSetVector (mask_w_mat_size, sizeof(*mask_w_mat), mask_w_mat, 1, mask_w_mat_dev, 1);
  if (stat != HIPBLAS_STATUS_SUCCESS)
  {
    printf ("mask_w_mat_dev download failed, status: %d, element size: %ld, total mem: %ld B, size:%d\n", stat,sizeof(*mask_w_mat), mask_w_mat_size*sizeof(*mask_w_mat),map_a*mask_side);
    hipFree (mask_w_mat_dev);
    hipblasDestroy(handle);
    return EXIT_FAILURE;
  }
  //printf ("mask_w_mat_dev download succeed\n");
  //printMaskWMat(mask_w_mat, mask_side, map_a);
  stat = hipblasGetVector (mask_w_mat_size, sizeof(*mask_w_mat), mask_w_mat_dev, 1, mask_w_mat, 1);
  if (stat != HIPBLAS_STATUS_SUCCESS)
  {
    printf ("mask_w_mat upload failed, status: %d, element size: %ld, total mem: %ld B, size:%d\n", stat,sizeof(*mask_w_mat), mask_w_mat_size*sizeof(*mask_w_mat),map_a*mask_side);
    showMemInfo();
    hipFree (mask_w_mat_dev);
    hipblasDestroy(handle);
    return EXIT_FAILURE;
  }
  //printf ("mask_w_mat_dev upload succeed\nprint mask_w_mat again\n");
  //printMaskWMat(mask_w_mat, mask_side, map_a);
  int particle_count = 0;
  int runs = 100;
  for(int pidx_x = 0; pidx_x < map_x; pidx_x++)
  for(int pidx_y = 0; pidx_y < map_y; pidx_y++)
  for(int pidx_a = 0; pidx_a < map_a; pidx_a++)
  {
    if(particle_count >= runs)
    {
      pidx_x = map_x;//break outter loop
      pidx_y = map_y;//break outter loop
      break;
    }
    particle_count++;
    //For each particle
    int pidx = IDX2G(pidx_a, pidx_x, pidx_y, map_x, map_y);
    //TODO get ngb_pre_w_vec with active position
    //TODO cp ngb_pre_w_vec to device
    for(int mask_a = 0; mask_a < map_a ; mask_a++)
    {
      for(int mask_x = 0 ; mask_x < mask_side ; mask_x++)
      {
        for(int mask_y = 0; mask_y < mask_side ; mask_y++)
        {
          int ngb_pidx_x = pidx_x - (mask_side-1)/2 + mask_x;
          int ngb_pidx_y = pidx_y - (mask_side-1)/2 + mask_y;
          int ngb_pidx_a = mask_a;
          int ngb_pidx = IDX2G(ngb_pidx_a, ngb_pidx_x, ngb_pidx_y, map_x, map_y);
          int ngb_pre_w_vec_idx = IDX2G(mask_a,mask_x,mask_y,mask_side,mask_side);
          if(ngb_pre_w_vec_idx < 0 || ngb_pre_w_vec_idx > mask_side*mask_side*map_a)
            printf("wrong ngb_pre_w_vec_idx %d from %d %d %d\n",ngb_pre_w_vec_idx, mask_a, mask_x, mask_y);
          //printf("(%d, %d, %d)=%d->%d",mask_x,mask_y,mask_a,ngb_pre_w_vec_idx,ngb_pidx);
          if(ngb_pidx_x < 0 || ngb_pidx_x > map_x || 
             ngb_pidx_y < 0 || ngb_pidx_y > map_y || 
             ngb_pidx_a < 0 || ngb_pidx_a > map_a)
          {
            //printf(" skipped \n");
            ngb_pre_w_vec[ngb_pre_w_vec_idx] = 0.0;
          }
          else
          {
            //printf("\n");
            ngb_pre_w_vec[ngb_pre_w_vec_idx] = pre_w[ngb_pidx];
            //printf(" = %f, %f\n",ngb_pre_w_vec[ngb_pre_w_vec_idx], pre_w[ngb_pidx]);
          }
        }
      }
    }
    //printf("setting ngb_pre_w_vec\n");
    ros::Time cuda11 = ros::Time::now();
    stat = hipblasSetVector(ngb_pre_w_vec_size , sizeof(*ngb_pre_w_vec), ngb_pre_w_vec, 1, ngb_pre_w_vec_dev, 1);
    ros::Time cuda12 = ros::Time::now();
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
      printf ("ngb_pre_w_vec data download failed\n");
      hipFree (ngb_pre_w_vec_dev);
      hipFree (mask_w_mat_dev);
      hipFree (cur_w_dev);
      hipblasDestroy(handle);
      return EXIT_FAILURE;
    }
    stat = hipblasGetVector(ngb_pre_w_vec_size , sizeof(*ngb_pre_w_vec_bak), ngb_pre_w_vec_dev, 1, ngb_pre_w_vec_bak, 1);
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
      printf ("ngb_pre_w_vec_bak data upload failed\n");
      return EXIT_FAILURE;
    }
    if(check(ngb_pre_w_vec, ngb_pre_w_vec_bak, ngb_pre_w_vec_size) != 0)
    {
      printNgbPreWVec(ngb_pre_w_vec, mask_side, map_a);
      printNgbPreWVec(ngb_pre_w_vec_bak, mask_side, map_a);
      printf("ngb_pre_w_vec and ngb_pre_w_vec_bak are not the same");
      return EXIT_FAILURE;
    }
    //get origin_particle_angle_idx
    int origin_particle_angle_idx = ((pidx+1)%map_a)*ngb_pre_w_vec_size;//assume this value instead of ANG2IDX(sample[pidx].v[2])
    //printf("executing hipblasDdot\n");
    /* A*B=W
    ngb_pre_w_vec_dev        : A vector 
    mask_w_mat_dev           : a matrix consisting of all B vectors, where the number of B vectors is 
    mask_w_mat_size          : the number of all elements in the matrix
    origin_particle_angle_idx: the index of each B vector in the matrix
    cur_w_dev                : an array for storing a sequence of W
    pidx                     : the index of cur_w_dev indicating which position should be used for storing a result W
    */
    ros::Time cuda21 = ros::Time::now();
    hipblasDdot(handle, ngb_pre_w_vec_size, ngb_pre_w_vec_dev, 1, (mask_w_mat_dev+origin_particle_angle_idx), 1, (cur_w_dev+pidx));
    ros::Time cuda22 = ros::Time::now();

    //vector B
    stat = hipblasGetVector(ngb_pre_w_vec_size , sizeof(*mask_w_vec_bak), (mask_w_mat_dev+origin_particle_angle_idx), 1, mask_w_vec_bak, 1);
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
      printf("upload mask_w_mat_dev+origin_particle_angle_idx failed\n");
      return EXIT_FAILURE;
    }
    ros::Time cuda31 = ros::Time::now();
    stat = hipblasGetVector(1 , sizeof(*single_w), (cur_w_dev+pidx), 1, single_w, 1);
    ros::Time cuda32 = ros::Time::now();
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
      printf("upload  failed\n");
      return EXIT_FAILURE;
    }
    ros::Time cpu11 = ros::Time::now();
    double cpu_w = CPUdot( ngb_pre_w_vec_bak, mask_w_vec_bak, ngb_pre_w_vec_size);
    ros::Time cpu12 = ros::Time::now();
    if(cpu_w != *single_w)
    {
      printf("print vector A\n");
      printNgbPreWVec(ngb_pre_w_vec_bak, mask_side, map_a);
      printf("print vector B\n");
      printNgbPreWVec(mask_w_vec_bak, mask_side, map_a);
      printf("result W from GPU is %f\n",*single_w);
      printf("result W from CPU is %f\n", cpu_w);
      printf("results from CPU and GPU are not the same\n");
      return EXIT_FAILURE;
    }
    //else
    //  printf("results from CPU and GPU are the same\n");
    int cuda1 = (cuda12 - cuda11).toNSec();
    int cuda2 = (cuda22 - cuda21).toNSec();
    int cuda3 = (cuda32 - cuda31).toNSec();
    int cudaall = (cuda1 + cuda2 + cuda3);
    int cpu = (cpu12 - cpu11).toNSec();
    vec_cuda1.push_back(cuda1);
    vec_cuda2.push_back(cuda2);
    vec_cuda3.push_back(cuda3);
    vec_cudaall.push_back(cudaall);
    vec_cpu.push_back(cpu);
    //printf("GPU times: %ld + %ld + %ld = %ld\n", cuda1, cuda2, cuda3, cudaall);
    //printf("CPU dot time: %ld\n", cpu);
  }
  //printf("original cur_w\n");
  //printWG(cur_w, map_x, map_y, map_a);
  hipblasGetVector(map_x*map_y*map_a, sizeof(*cur_w_dev), cur_w_dev, 1, cur_w, 1);
  //printf("cur_w\n");
  //printWG(cur_w, map_x, map_y, map_a);

  printf("size of computation: %ld\n", vec_cpu.size());
  printf("printing GPU mem info\n");
  showMemInfo();
  hipFree (mask_w_mat_dev);
  hipFree (ngb_pre_w_vec_dev);
  hipFree (cur_w_dev);
  free(pre_w);
  free(cur_w);
  free(mask_w_mat);
  free(ngb_pre_w_vec);
  printf("printing GPU mem info after releasing\n");
  showMemInfo();
  printf("cuda 1 avg time  : %f ns\n", 1.0*std::accumulate(vec_cuda1.begin(), vec_cuda1.end(), 0)/vec_cuda1.size());
  printf("cuda 2 avg time  : %f ns\n", 1.0*std::accumulate(vec_cuda2.begin(), vec_cuda2.end(), 0)/vec_cuda2.size());
  printf("cuda 3 avg time  : %f ns\n", 1.0*std::accumulate(vec_cuda3.begin(), vec_cuda3.end(), 0)/vec_cuda3.size());
  printf("cuda all avg time: %f ns\n", 1.0*std::accumulate(vec_cudaall.begin(), vec_cudaall.end(), 0)/vec_cudaall.size());
  printf("cpu avg time     : %f ns\n", 1.0*std::accumulate(vec_cpu.begin(), vec_cpu.end(), 0)/vec_cpu.size());
  return EXIT_SUCCESS;
}

void printNgbPreWVec(double*& ngb_pre_w_vec, int mask_side, int map_a)
{
  for(int mask_a = 0 ; mask_a < map_a;mask_a++)
  {
  for(int mask_x = 0 ; mask_x < mask_side;mask_x++)
    {
  for(int mask_y = 0 ; mask_y < mask_side;mask_y++)
      {
        int ngb_pre_w_vec_idx = IDX2G(mask_a,mask_x,mask_y,mask_side,mask_side);
        printf ("%f ", ngb_pre_w_vec[ngb_pre_w_vec_idx]);
      }
      printf("\n");
    }
    printf("\n");
  }
  printf("\n");
}

int allocateNgbPreWVec(double*& ngb_pre_w_vec, int mask_side, int map_a,size_t total_size)
{
  ngb_pre_w_vec = (double *)malloc( total_size*sizeof(*ngb_pre_w_vec) );
  if (!ngb_pre_w_vec )
  {
    printf ("host memory allocation failed\n");
    return 1;
  }
  return 0;
}

int allocateMaskWMat(double*& mask_w_mat, int mask_side, int map_a,size_t total_size)
{
  mask_w_mat = (double*)malloc( total_size*sizeof(*mask_w_mat) );
  if (!mask_w_mat )
  {
    printf ("host memory allocation failed\n");
    return 1;
  }
  int h,i,j,k;
  for (h = 0; h < map_a; h++)
    for (i = 0; i < mask_side; i++)
      for (j = 0; j < mask_side; j++)
        for (k = 0; k < map_a; k++)
          //mask_w_mat[IDX2Q(h,k,i,j,map_a,mask_side,mask_side)] = (double)(IDX2Q(h,k,i,j,map_a,mask_side,mask_side));
          mask_w_mat[IDX2Q(h,k,i,j,map_a,mask_side,mask_side)] = 2;
  return 0;
}

void printMaskWMat(double* mask_w_mat, int mask_side, int map_a)
{
  int h,i,j,k;
  printf("mask_w_mat\n");
  for (h = 0; h < map_a; h++)
  {
    for (i = 0; i < mask_side; i++)
    {
      for (j = 0; j < mask_side; j++)
      {
        for (k = 0; k < map_a; k++)
        {
          printf ("%7.0f", mask_w_mat[IDX2Q(h,k,i,j,map_a,mask_side,mask_side)]);
        }
        printf("\n");
      }
      printf("\n");
    }
    printf("\n");
  }
}

void printWG(double* pre_w, int map_x, int map_y, int map_a)
{
  int i,j,k;
  for (k = 0; k < map_a; k++)
  {
    for (i = 0; i < map_x; i++)
    {
      for (j = 0; j < map_y; j++)
      {
        printf ("%f ", pre_w[IDX2G(k,i,j,map_x,map_y)]);
      }
      printf("\n");
    }
    printf("\n");
  }
  printf("\n");
  ;
}

int allocatePreW(double*& pre_w, int map_x, int map_y, int map_a)
{
  pre_w = (double *)malloc( map_x*map_y*map_a*sizeof(*pre_w));
  if(!pre_w)
  {
    printf("host memory allocation failed\n");
    return 1;
  }
  int i,j,k;
  for (k = 0; k < map_a; k++)
    for (j = 0; j < map_y; j++)
      for (i = 0; i < map_x; i++)
        //pre_w[IDX2G(k,i,j,map_x,map_y)] = (double)IDX2G(k,i,j,map_x,map_y);
        pre_w[IDX2G(k,i,j,map_x,map_y)] = 1;
  //printf("pre_w\n");
  //printWG(pre_w, map_x, map_y, map_a);
  return 0;
}
